#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <iostream>

// includes CUDA Runtime
#include <hip/hip_runtime.h>

using namespace std;

// 两个向量加法kernel，grid和block均为一维
__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长stride是整个grid的线程数,有时候向量的元素数很多，这时候可以将在每个线程实现多个元素（元素总数/线程总数）的加法，相当于使用了多个grid来处理
    // 不过下面的例子一个线程只处理一个元素，所以kernel里面的循环是不执行的
    int stride = blockDim.x * gridDim.x;
    // if (threadIdx.x == 0) 
    //     printf("Hello thread %d, stride=%f\n", threadIdx.x, stride) ;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

int main()
{
    // 向量长度
    int N = 1 << 20;
    int nBytes = N * sizeof(float);
    // 申请host内存
    float *x, *y, *z;
    x = (float*)malloc(nBytes);
    y = (float*)malloc(nBytes);
    z = (float*)malloc(nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 申请device内存
    float *d_x, *d_y, *d_z;
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_z, nBytes);

    // 将host数据拷贝到device
    hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add << < gridSize, blockSize >> >(d_x, d_y, d_z, N);

    // 将device得到的结果拷贝到host
    hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(z[i] - 30.0));
        std::cout<<"z["<<i<<"]="<<z[i]<<std::endl;
    }
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放device内存
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    // 释放host内存
    free(x);
    free(y);
    free(z);

    return 0;
}